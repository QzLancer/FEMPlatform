#include "hip/hip_runtime.h"
#include "FEM2DNDDRCUDASolver.cuh"

#include <time.h>
#include <stdio.h>

__managed__ double error = 0;

void FEM2DNDDRCUDASolver::solve()
{
	//CPU�����ʼ������Ȼ�����ݿ�����GPU��
	processBoundaryCondition();
	processLoad();
	processNDDRNode();
	makeTrangle();
	processMaterial();

	GPUInitialMallocCopy();

	//makeTrangleinDevice << <CudaBlckNum, CudaThrdNum >> > (m_num_nodes, d_mp_node, d_mp_triele);
	//hipDeviceSynchronize();


	double* a, * b;
	hipMalloc(&a, m_num_nodes * sizeof(double));
	hipMalloc(&b, m_num_nodes * sizeof(double));
	error = 0;
	for (int iter = 0; iter < maxitersteps; ++iter) {
		nodeAnalysis << <CudaBlckNum, CudaThrdNum >> > (m_num_nodes, d_mp_node, d_mp_triele);
		calculateGlobalError << <CudaBlckNum, CudaThrdNum >> > (m_num_nodes, d_mp_node, d_mp_triele, a, b);
		hipDeviceSynchronize();

		if ((iter + 1) % 100 == 0) {
			cout << "Iteration step: " << iter + 1 << ", Relative error: " << error << endl;
		}
		if (error > maxerror) {
			copyAttoAtold << <CudaBlckNum, CudaThrdNum >> > (m_num_nodes, d_mp_node);
		}
		else {
			cout << "Iteration step: " << iter + 1 << endl;
			cout << "Nonlinear NDDR iteration finish.\n";
			break;
		}
	}

	hipMemcpy(mp_node, d_mp_node, m_num_nodes * sizeof(CNode), hipMemcpyDeviceToHost);

	hipFree(b);
	hipFree(a);
	GPUFree();
}

FEM2DNDDRCUDASolver::~FEM2DNDDRCUDASolver()
{
	//if (mp_triele != nullptr) {
	//	hipFree(mp_triele);
	//	mp_triele = nullptr;
	//}
	//if (mp_edgele != nullptr) {
	//	hipFree(mp_edgele);
	//	mp_edgele = nullptr;
	//}
	//if (mp_vtxele != nullptr) {
	//	hipFree(mp_vtxele);
	//	mp_vtxele = nullptr;
	//}
	//if (mp_node != nullptr) {
	//	hipFree(mp_node);
	//	mp_node = nullptr;
	//}
}

//void FEM2DNDDRCUDASolver::setNodes(const int _numofnodes, CNode* const _nodes)
//{
//	cout << "FEM2DNDDRCUDASolver::setNodes\n";
//	m_num_nodes = _numofnodes;
//	//hipMallocManaged((void**)&mp_node, m_num_nodes * sizeof(CNode));
//	//memcpy(mp_node, _nodes, m_num_nodes * sizeof(CNode));
//	hipMalloc(&mp_node, m_num_nodes * sizeof(CNode));
//	hipMemcpy(mp_node, _nodes, m_num_nodes * sizeof(CNode), hipMemcpyHostToDevice);
//}
//
//void FEM2DNDDRCUDASolver::setVtxElements(const int _numofvtx, CVtxElement* const _vtxele)
//{
//	m_num_vtxele = _numofvtx;
//	//hipMallocManaged((void**)&mp_vtxele, m_num_vtxele * sizeof(CVtxElement));
//	//memcpy(mp_vtxele, _vtxele, m_num_vtxele * sizeof(CVtxElement));
	//hipMalloc(&mp_vtxele, m_num_vtxele * sizeof(CVtxElement));
	//hipMemcpy(mp_vtxele, _vtxele, m_num_vtxele * sizeof(CVtxElement), hipMemcpyHostToDevice);
//}
//
//void FEM2DNDDRCUDASolver::setEdgElements(const int _numofedg, CEdgElement* const _edgele)
//{
//	m_num_edgele = _numofedg;
//	//hipMallocManaged((void**)&mp_edgele, m_num_edgele * sizeof(CEdgElement));
//	//memcpy(mp_edgele, _edgele, m_num_edgele * sizeof(CEdgElement));
//	hipMalloc((void**)&mp_edgele, m_num_edgele * sizeof(CEdgElement));
//	hipMemcpy(mp_edgele, _edgele, m_num_edgele * sizeof(CEdgElement), hipMemcpyHostToDevice);
//}
//
//void FEM2DNDDRCUDASolver::setTriElements(const int _numoftri, CTriElement* const _triele)
//{
//	cout << "FEM2DNDDRCUDASolver::setTriElements(const int _numoftri, CTriElement* const _triele)\n";
//	m_num_triele = _numoftri;
//	//hipMallocManaged((void**)&mp_triele, m_num_triele * sizeof(CTriElement));
//	//memcpy(mp_triele, _triele, m_num_triele * sizeof(CTriElement));
	//hipMalloc((void**)&mp_triele, m_num_triele * sizeof(CTriElement));
	//hipMemcpy(mp_triele, _triele, m_num_triele * sizeof(CTriElement), hipMemcpyHostToDevice);
//}

//��GPU�д���BH����
//��GPU�д����������飬���ϵ�BH����ָ��GPU
//Ŀǰ�Ǹ���domain�Ĵ�С��������Ͽռ䣬���н�һ���Ż��Ŀ���
void FEM2DNDDRCUDASolver::processMaterial()
{
	//for (int i_tri = 0; i_tri < m_num_triele; ++i_tri) {
	//	int domain = mp_triele[i_tri].domain;
	//	cout << "domain: " << domain << endl;
	//	FEMMaterial* material = materialmap[domain];
	//	hipMallocManaged(&mp_triele[i_tri].material, sizeof(FEMMaterial));
	//	mp_triele[i_tri].material->GPUCopy(*material);
	//}
	int domainsize = 0;
	for (auto mat = materialmap.begin(); mat != materialmap.end(); mat++ ) {
		if (domainsize < mat->first) {
			domainsize = mat->first;
		}
	}

	//��CPU�д�����������
	cout << "domainsize: " << domainsize << endl;
	materialarray = new FEMMaterial[domainsize];

	for (int i = 0; i < domainsize; ++i) {
		materialarray[i].GPUCopy(*materialmap[i + 1]);
	}

	//���������鿽����GPU��
	hipMalloc(&d_materialarray, domainsize * sizeof(FEMMaterial));
	hipMemcpy(d_materialarray, materialarray, domainsize * sizeof(FEMMaterial), hipMemcpyHostToDevice);

	//delete[] materialarray;
	
	//��material�󶨵�TriElement��
	//��ΪGPU�汾�����ǵ�processLoad��Boundary��ͨ���ԣ����߶�����GPU�ڴ濽��֮ǰ,�����ʹ��CPU�汾��
	//GPU�汾�У�GPUInitialMallocCopy()����processMaterial()֮ǰ
	//GPU�汾
	//assignMattoTriEle << <CudaBlckNum, CudaThrdNum >> > (m_num_triele, d_mp_triele, d_materialarray);
	//hipDeviceSynchronize();
 
	//��ΪCPU�汾
	//CPU�汾�У�GPUInitialMallocCopy()����processMaterial()֮��
	for (int i_tri = 0; i_tri < m_num_triele; ++i_tri) {
		int matid = mp_triele[i_tri].domain - 1;
		mp_triele[i_tri].material = &d_materialarray[matid];
	}

	//materialArrayTest << <CudaBlckNum, CudaThrdNum >> > (domainsize, d_materialarray);
	//hipDeviceSynchronize();
}

void FEM2DNDDRCUDASolver::GPUFree()
{
	hipFree(d_mp_node);
	hipFree(d_mp_triele);
	hipFree(d_materialarray);
	delete[] materialarray;
}

void FEM2DNDDRCUDASolver::GPUInitialMallocCopy()
{
	//CUDA initialize
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	std::cout << "Number of device: " << num_devices << endl;
	if (num_devices > 1) {
		int max_multiprocessors = 0, max_device = 0;
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
	}
	for (int i = 0; i < num_devices; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	hipSetDevice(0);
	
	//����ڵ��TriElement���ڴ�
	hipMalloc(&d_mp_node, m_num_nodes * sizeof(CNode));
	hipMemcpy(d_mp_node, mp_node, m_num_nodes * sizeof(CNode), hipMemcpyHostToDevice);
	hipMalloc(&d_mp_triele, m_num_triele * sizeof(CTriElement));
	hipMemcpy(d_mp_triele, mp_triele, m_num_triele * sizeof(CTriElement), hipMemcpyHostToDevice);

}

void FEM2DNDDRCUDASolver::processNDDRNode()
{
	for (int i_tri = 0; i_tri < m_num_triele; ++i_tri) {
		for (int j = 0; j < 3; ++j) {
			int n = mp_triele[i_tri].n[j];
			int id = mp_node[n].NumberofNeighbourElement;
			mp_node[n].NeighbourElementId[id] = i_tri;
			mp_node[n].NeighbourElementNumber[id] = j;
			mp_node[n].NumberofNeighbourElement++;
			//printf("global n:%d, NumberofNeighbourElement:%d\n", n, mp_node[n].NumberofNeighbourElement);
		}
		//printf("global ele:%d, J:%f\n", i_tri, mp_triele[i_tri].J);
	}
}

__global__ void nodeAnalysis(int d_m_num_nodes, CNode* d_mp_node, CTriElement* d_mp_triele)
{
	int n = threadIdx.x + blockIdx.x * blockDim.x;
	if (n >= d_m_num_nodes)
		return;
	if (d_mp_node[n].bdr == 1)
		return;
	//�ڵ��ڲ���������
	int maxNRitersteps = 1;
	double Ati = 0;
	for (int NRiter = 0; NRiter < maxNRitersteps; ++NRiter) {
		double S = 0, F = 0;
		double J = 0, Fj = 0;
		//װ�����
		for (int k = 0; k < d_mp_node[n].NumberofNeighbourElement; ++k) {
			int i_tri = d_mp_node[n].NeighbourElementId[k];
			CTriElement triele = d_mp_triele[i_tri];
			int nodenumber = d_mp_node[n].NeighbourElementNumber[k];
			double mu = triele.material->getMuinDevice(d_mp_triele[i_tri].B);
			double mut = triele.material->getMuinDevice(d_mp_triele[i_tri].B) * triele.xdot;
			//printf("nodeid: %d, nodenumber: %d, mut: %f\n",n, nodenumber, mut);
			//printf("triele.j: %f\n", triele.J);
			//�������Ե�Ԫ
			if (triele.material->getLinearFlaginDevice() == true) {
				for (int i = 0; i < 3; ++i) {
					double Se = triele.C[nodenumber][i] / mut;
					if (nodenumber == i) {
						S += Se;
						F += triele.J * triele.area / 3;
					}
					else {
						F -= Se * d_mp_node[triele.n[i]].At_old;
					}
				}
			}
			//��������Ե�Ԫ
			else {
				double dvdb, dvdbt, Bt, sigmai = 0, sigmaj = 0;
				dvdb = triele.material->getdvdBinDevice(d_mp_triele[i_tri].B);
				dvdbt = dvdb / triele.xdot / triele.xdot;
				Bt = d_mp_triele[i_tri].B * triele.xdot;
				for (int i = 0; i < 3; ++i) {
					for (int m = 0; m < 3; ++m) {
						if (m == nodenumber) {
							sigmai += triele.C[i][m] * Ati;
						}
						else {
							sigmai += triele.C[i][m] * d_mp_node[triele.n[m]].At_old;
						}
					}
					for (int j = 0; j < 3; ++j) {
						for (int m = 0; m < 3; ++m) {
							if (m == nodenumber) {
								sigmaj += triele.C[j][m] * Ati;
							}
							else {
								sigmaj += triele.C[j][m] * d_mp_node[triele.n[m]].At_old;
							}
						}
					}
				}
				for (int i = 0; i < 3; ++i) {
					if (Bt != 0) {
						J = triele.C[nodenumber][i] / mut + sigmai * sigmaj / Bt / triele.area;
					}
					else {
						J = triele.C[nodenumber][i] / mut;
					}
					if (nodenumber == i) {
						S += J;
						F += (J - triele.C[nodenumber][i] / mut) * Ati;
					}
					else {
						F += (J - triele.C[nodenumber][i] / mut) * d_mp_node[triele.n[i]].At_old;
						F -= J * d_mp_node[triele.n[i]].At_old;
					}
				}
			}
		}
		//printf("NR_iter: %d\n", NRiter);
		//if (F != 0)
		//	printf("NR_iter: %d, nodeid: %d, S: %f, F: %f\n", NRiter, n, S, F);
		//Ati��ʵ�ϲ�ȫ��Ϊ0�������޷��������С�����λ��
		Ati = F / S;
		//if (Ati != 0) {
		//	printf("NRiter: %d, nodeid: %d, S: %f, F: %f, Ati: %f\n", NRiter, n, S, F, Ati);
		//}
		//NR�����������ж�
		double a = (Ati - d_mp_node[n].At) * (Ati - d_mp_node[n].At);
		double b = Ati * Ati;
		double NRerror = sqrtf(a) / sqrtf(b);
		//printf("Ati: %f, d_mp_node[n].At: %f, NRerror: %f\n", Ati, d_mp_node[n].At, NRerror);
		//__syncthreads();
		if (Ati == 0) {
			continue;
		}
		if (NRerror > 1e-5) {
			d_mp_node[n].At = Ati;
			d_mp_node[n].A = d_mp_node[n].At / d_mp_node[n].x;
			for (int i = 0; i < d_mp_node[n].NumberofNeighbourElement; ++i) {
				//updateB
				double bx = 0, by = 0;
				int i_tri = d_mp_node[n].NeighbourElementId[i];
				for (int j = 0; j < 3; ++j) {
					int n = d_mp_triele[i_tri].n[j];
					bx += d_mp_triele[i_tri].R[j] * d_mp_node[n].A;
					by += d_mp_triele[i_tri].Q[j] * d_mp_node[n].A;
				}
				bx = bx / 2 / d_mp_triele[i_tri].area;
				d_mp_triele[i_tri].Bx = bx;
				by = -by / 2 / d_mp_triele[i_tri].area;
				d_mp_triele[i_tri].By = by;
				d_mp_triele[i_tri].B = sqrtf(bx * bx + by * by);
			}
		}
		else {
			//printf("n:%d, NRiter: %d\n", n, NRiter);
			break;
		}
	}
}

__global__ void calculateGlobalError(int d_m_num_nodes, CNode* d_mp_node, CTriElement* d_mp_triele, double* a, double* b)
{
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	if (n >= d_m_num_nodes) {
		return;
	}

	a[n] = (d_mp_node[n].At - d_mp_node[n].At_old) * (d_mp_node[n].At - d_mp_node[n].At_old);
	b[n] = d_mp_node[n].At * d_mp_node[n].At;
	__syncthreads();

	//a��b��Լ���
	int leng = d_m_num_nodes;
	for (int i = d_m_num_nodes / 2.0 + 0.5; i > 1; i = i / 2.0 + 0.5) {
		if (n < i)
		{

			if (n + i < leng)
			{
				a[n] += a[n + i];
				b[n] += b[n + i];
			}
		}
		__syncthreads();
		leng = leng / 2.0 + 0.5;
	}

	if (n == 0) {
		a[0] = a[0] + a[1];
		b[0] = b[0] + b[1];
		error = sqrtf(a[0]) / sqrtf(b[0]);
	}
}

__global__ void copyAttoAtold(int d_m_num_nodes, CNode* d_mp_node)
{
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	if (n >= d_m_num_nodes) {
		return;
	}

	d_mp_node[n].At_old = d_mp_node[n].At;
}

__global__ void makeTrangleinDevice(int numofTrangle, CNode* d_mp_node, CTriElement* d_mp_triele)
{
	//��ô�жϱ�֤thread������numofTrangle?
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index >= numofTrangle)
		return;

	int k, m, n;
	double q0, q1, q2, r0, r1, r2, area;
	k = d_mp_triele[index].n[0];
	m = d_mp_triele[index].n[1];
	n = d_mp_triele[index].n[2];

	q0 = d_mp_node[m].y - d_mp_node[n].y;
	d_mp_triele[index].Q[0] = q0;
	q1 = d_mp_node[n].y - d_mp_node[k].y;
	d_mp_triele[index].Q[1] = q1;
	q2 = d_mp_node[k].y - d_mp_node[m].y;
	d_mp_triele[index].Q[2] = q2;

	r0 = d_mp_node[n].x - d_mp_node[m].x;
	d_mp_triele[index].R[0] = r0;
	r1 = d_mp_node[k].x - d_mp_node[n].x;
	d_mp_triele[index].R[1] = r1;
	r2 = d_mp_node[m].x - d_mp_node[k].x;
	d_mp_triele[index].R[2] = r2;

	area = 0.5 * std::abs(q1 * r2 - r1 * q2);
	d_mp_triele[index].area = area;

	d_mp_triele[index].rc = (d_mp_node[k].x +
		d_mp_node[m].x +
		d_mp_node[n].x) / 3;
	d_mp_triele[index].zc = (d_mp_node[k].y +
		d_mp_node[m].y +
		d_mp_node[n].y) / 3;

	int flag = 0;
	for (int f = 0; f < 3; f++) {
		if (d_mp_node[d_mp_triele[index].n[f]].x < 1e-7) {
			flag++;
		}
	}

	//�������������İ뾶
	if (flag == 2) {
		d_mp_triele[index].xdot = d_mp_triele[index].rc;
	}
	else {
		d_mp_triele[index].xdot = 1 / (d_mp_node[k].x + d_mp_node[m].x);
		d_mp_triele[index].xdot += 1 / (d_mp_node[k].x + d_mp_node[n].x);
		d_mp_triele[index].xdot += 1 / (d_mp_node[m].x + d_mp_node[n].x);
		d_mp_triele[index].xdot = 1.5 / d_mp_triele[index].xdot;
	}

	//����һ����������ԳƵ�Ԫϵ������
	for (int i = 0; i < 3; ++i) {
		for (int j = 0; j < 3; ++j) {
			d_mp_triele[index].C[i][j] = ((d_mp_triele[index].R[i] * d_mp_triele[index].R[j] + d_mp_triele[index].Q[i] * d_mp_triele[index].Q[j])) / (4 * d_mp_triele[index].area);
		}
	}
}

__global__ void assignMattoTriEle(int numofTrangle, CTriElement* d_mp_triele, FEMMaterial* d_materialarray)
{
	int i_tri = blockDim.x * blockIdx.x + threadIdx.x;
	if (i_tri >= numofTrangle) {
		return;
	}

	int matid = d_mp_triele[i_tri].domain - 1;
	d_mp_triele[i_tri].material = &d_materialarray[matid];
}
